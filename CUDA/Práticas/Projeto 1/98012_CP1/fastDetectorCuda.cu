#include "hip/hip_runtime.h"
// Based on CUDA SDK template from NVIDIA

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <unistd.h>
#include <assert.h>
#include <float.h>

// includes, project
#include <hip/hip_runtime_api.h>
#include <helper_image.h>

#define max(a,b) (((a)>(b))?(a):(b))
#define min(a,b) (((a)<(b))?(a):(b))

#define MAX_BRIGHTNESS 255
#define FADEDIV 2

// pixel base type
// Use int instead `unsigned char' so that we can
// store negative values.
typedef int pixel_t;

__managed__ int offset[16]; // offsets to circle pixels

// initializes global array of offsets of neighborhood pixels in circle of radius 3
void makeOffsets(int offset[], const int row_stride)
{
    offset[0] = 0 + row_stride * 3;
    offset[1] = 1 + row_stride * 3;
    offset[2] = 2 + row_stride * 2;
    offset[3] = 3 + row_stride * 1;
    offset[4] = 3 + row_stride * 0;
    offset[5] = 3 + row_stride * -1;
    offset[6] = 2 + row_stride * -2;
    offset[7] = 1 + row_stride * -3;
    offset[8] = 0 + row_stride * -3;
    offset[9] = -1 + row_stride * -3;
    offset[10] = -2 + row_stride * -2;
    offset[11] = -3 + row_stride * -1;
    offset[12] = -3 + row_stride * 0;
    offset[13] = -3 + row_stride * 1;
    offset[14] = -2 + row_stride * 2;
    offset[15] = -1 + row_stride * 3;
}

#define DARKER (-1)
#define SIMILAR 0
#define BRIGHTER 1

// detects if pixel pointed to by h_ipixel is a FAST feature:
//   has at least th_count consecutive neighbours (along circle radius 3)
//   that are darker or brighter by th_diff
__host__ __device__ int fastCorner(const pixel_t *h_ipixel,
               const int w, const int h,
               const int th_count, // min count to detect corners size
               const int th_diff   // threshold diff to count
)
{
    int consec = 0;
    int dk_consec = 0, br_consec = 0;
    int dk_begin = 0, dk_begin_count = 0;
    int br_begin = 0, br_begin_count = 0;

    pixel_t pix_val = *h_ipixel;
    int intensity = SIMILAR;

    int p;
    for (p = 0; p < 16; p++)
    {
        if (h_ipixel[offset[p]] < pix_val - th_diff)
        { // Darker neighbor
            if (p == 0)
            {
                dk_begin = 1;
            }
            if (intensity == DARKER)
            {
                consec++;
            }
            else
            {
                if (intensity == BRIGHTER && consec > br_consec)
                {
                    if (br_begin == 1)
                    {
                        br_begin_count = consec;
                        br_begin = 0;
                    }
                    br_consec = consec;
                }
                consec = 1;
            }
            intensity = DARKER;
        }

        else if (h_ipixel[offset[p]] > pix_val + th_diff)
        { // Brighter neighbor
            if (p == 0)
            {
                br_begin = 1;
            }
            if (intensity == BRIGHTER)
            {
                consec++;
            }
            else
            {
                if (intensity == DARKER && consec > dk_consec)
                {
                    if (dk_begin == 1)
                    {
                        dk_begin_count = consec;
                        dk_begin = 0;
                    }
                    dk_consec = consec;
                }
                consec = 1;
            }
            intensity = BRIGHTER;
        }
        else
        { // Similar Neighbor
            if (intensity == DARKER && consec > dk_consec)
            {
                if (dk_begin == 1)
                {
                    dk_begin_count = consec;
                    dk_begin = 0;
                }
                dk_consec = consec;
            }
            if (intensity == BRIGHTER && consec > br_consec)
            {
                if (br_begin == 1)
                {
                    br_begin_count = consec;
                    br_begin = 0;
                }
                br_consec = consec;
            }
            consec = 0;
            intensity = SIMILAR;
        }
    }

    if (intensity == DARKER)
    {
        if (dk_begin_count)
        { // merge consecutive pixels
            if (consec + dk_begin_count > dk_consec)
                dk_consec = consec + dk_begin_count;
        }
        else if (consec > dk_consec)
        {
            dk_consec = consec;
        }
    }

    if (intensity == BRIGHTER)
    {
        if (br_begin_count)
        { // merge consecutive pixels
            if (consec + br_begin_count > br_consec)
                br_consec = consec + br_begin_count;
        }
        else if (consec > br_consec)
        {
            br_consec = consec;
        }
    }
    if (dk_consec >= th_count || br_consec >= th_count)
    {
        return 1;
    }
    return 0;
}

// returns the score of pixel pointed to by h_ipixel
__host__ __device__ int fastScore(const pixel_t *h_ipixel,
              const int w, const int h,
              const int th_count)
{
    int scoremin = 1;
    int scoremax = max(MAX_BRIGHTNESS-*h_ipixel,*h_ipixel);

    while (scoremax - scoremin > 1)
    {
        if (fastCorner(h_ipixel, w, h, th_count, (scoremin + scoremax) / 2))
        {
            scoremin = (scoremin + scoremax) / 2;
        }
        else {
            scoremax = (scoremin + scoremax) / 2;
        }
    }

    return scoremin;
}

// detects all FAST corners in image h_idata and marks them with MAX_BRIGHTNESS in h_odata
void fastDetectCorners(const pixel_t *h_idata,
                       const int w, const int h,
                       const int th_count, // min count to detect corners size
                       const int th_diff,  // threshold diff to count
                       pixel_t *h_odata)
{
    int i, j, count = 0;

    for (i = 3; i < h - 3; i++) // height image
    {
        for (j = 3; j < w - 3; j++) // width image
        {
            if (fastCorner(h_idata + i * w + j, w, h, th_count, th_diff))
            {
                h_odata[i * w + j] = MAX_BRIGHTNESS;
                count++;
            }
        }
    }
    printf("detected %d features\n", count);
}

// FAST non-maximum suppression
void nonMaximumSupression(const pixel_t *in, const pixel_t *corners,
                          pixel_t *nms,
                          const int w, const int h, const int th_count)
{
    int count = 0;

    int *corner_score = (int *) malloc(w*h * sizeof(int));

    // determine score of each corner
    for (int i = 0; i < h; i++)
    {
        for (int j = 0; j < w; j++)
        {
            const int c = i * w + j;
            if (corners[c] == MAX_BRIGHTNESS)
            {
                  corner_score[c] = fastScore(in + c, w, h, th_count);
            }
            else {
                    
                  corner_score[c] = 0;
            }
        }
    }

    // keep only corner with local maximum score
    for (int i = 0; i < h; i++)
    {
        for (int j = 0; j < w; j++)
        {
            const int c = i * w + j;
            if (corners[c] != MAX_BRIGHTNESS)
            {
                continue;
            }

            int score_c = corner_score[c];
            int score_n;

            // check all neighbors
            for(int ni = max(0,i-1); ni < min(h,i+2); ni++) {
                for(int nj = max(0,j-1); nj < min(w,j+2); nj++) {
                    if(ni == i && nj == j) continue;

                    int nc = ni * w + nj; 
                    score_n = corner_score[nc];
                    if (score_n >= score_c) {
                        nms[c] = 0;
                        goto next;
                    }
                }
            }

            nms[c] = MAX_BRIGHTNESS;
            count++;
            next: continue;
        }
    }

    free(corner_score);

    printf("nonmax features %d\n", count);
}


// fast detector code to run on the host
void fastDetectorHost(const pixel_t *h_idata, const int w, const int h,
                      const int th_count, // min count to detect corners size
                      const int th_diff,  // threshold diff to count
                      const bool nonmaxflag,
                      pixel_t *h_odata)
{
    int i, j; // indexes in image

    // initialize h_odata to zero
    memset(h_odata, 0, h*w*sizeof(pixel_t));

    makeOffsets(offset, w);

    // corner detection
    fastDetectCorners(h_idata, w, h, th_count, th_diff, h_odata);


    if (nonmaxflag)
    {
        pixel_t *aux = (pixel_t *)malloc(w * h * sizeof(pixel_t));

        memcpy(aux, h_odata, w * h * sizeof(pixel_t));

        //non max supression
        nonMaximumSupression(h_idata, aux, h_odata, w, h, th_count);

        free(aux);
    }

    //add faded original image as background
    for (i = 0; i < h; i++) // height image
    {
        for (j = 0; j < w; j++) // width image
        {
            if(h_odata[i * w + j]!=MAX_BRIGHTNESS) {
                h_odata[i * w + j] = h_idata[i * w + j] / FADEDIV; // to obtain a faded background image
            }
        }
    }
}


__global__ void fastDetectCorners_CUDA(const pixel_t *h_idata,
                       const int w, const int h,
                       const int th_count, // min count to detect corners size
                       const int th_diff,  // threshold diff to count
                       pixel_t *h_odata)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = y * w + x;

    if (x >= 3 && x < w - 3 && y >= 3 && y < h - 3)
    {
        if (fastCorner(h_idata + idx, w, h, th_count, th_diff))
        {
            h_odata[idx] = MAX_BRIGHTNESS;
        }
    }
}



__global__ void nonMaximumSupressionKernel_p1(pixel_t *in, pixel_t *corners,
                                            pixel_t *corner_score,
                                            const int w, const int h, const int th_count)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < w && y < h)
    {
        int idx = y * w + x;
        if (corners[idx] == MAX_BRIGHTNESS)
        {
            corner_score[idx] = fastScore(in + idx, w, h, th_count);
        }
        else
        {
            corner_score[idx] = 0;
        }
    }
}

__global__ void nonMaximumSupressionKernel_p2(pixel_t *in, pixel_t *corners,
                                            pixel_t *nms, pixel_t *corner_score,
                                            const int w, const int h, const int th_count)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = y * w + x;
    if (x < w && y < h && corners[idx] == MAX_BRIGHTNESS)
    {
        int score_c = corner_score[idx];
        int score_n;

        for (int ni = max(0, y - 1); ni < min(h, y + 2); ni++)
        {
            for (int nj = max(0, x - 1); nj < min(w, x + 2); nj++)
            {
                if (ni == y && nj == x)
                    continue;

                int nc = ni * w + nj;
                score_n = corner_score[nc];
                if (score_n >= score_c)
                {
                    nms[idx] = 0;
                    goto next;
                }
            }
        }

        nms[idx] = MAX_BRIGHTNESS;
        next: ;
    }
}

// fast detector code to run on the GPU - Global Memory
void fastDetectorDevice(const pixel_t *h_idata, const int w, const int h,
                      const int th_count, const int th_diff,
                      const bool nonmaxflag, pixel_t *h_odata)
{
    int size = w * h * sizeof(pixel_t);
    pixel_t *d_idata, *d_odata;
    int *dev_offsets;

    // Allocate GPU memory
    hipMalloc((void **)&d_idata, size);
    hipMalloc((void **)&d_odata, size);
    hipMalloc((void **)&dev_offsets, 16 * sizeof(int));

    // Copy input data from host to device
    hipMemcpy(d_idata, h_idata, size, hipMemcpyHostToDevice);
    hipMemcpy(d_odata, h_odata, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_offsets, offset, 16 * sizeof(int), hipMemcpyHostToDevice);

    // Calculate grid and block dimensions
    dim3 blockDim(16, 16);
    dim3 gridDim((w + blockDim.x - 1) / blockDim.x, (h + blockDim.y - 1) / blockDim.y);

    // Launch the kernel
    fastDetectCorners_CUDA<<<gridDim, blockDim>>>(d_idata, w, h, th_count, th_diff, d_odata);
    hipDeviceSynchronize(); // Wait for the GPU launched work to complete

    if (nonmaxflag) // If true
    {
        pixel_t *dev_aux;
        // Allocate GPU memory
        hipMalloc((void**)&dev_aux,size);
        // Copy dev_odata to dev_aux
        hipMemcpy(dev_aux,d_odata,size,hipMemcpyDeviceToDevice);

        pixel_t *corner_score;
        // Allocate GPU memory
        hipMalloc((void**)&corner_score,size);
      
        // Launch the nonMaximumSupression kernel
        nonMaximumSupressionKernel_p1<<<gridDim, blockDim>>>(d_idata, dev_aux, corner_score, w, h, th_count);
        hipDeviceSynchronize();
        nonMaximumSupressionKernel_p2<<<gridDim, blockDim>>>(d_idata, dev_aux, d_odata, corner_score, w, h, th_count);

        hipFree(dev_aux);
        hipFree(corner_score);
    }

    // copy output data from device to host
    hipMemcpy(h_odata, d_odata, size, hipMemcpyDeviceToHost);

    int i, j; // indexes in image
    //add faded original image as background
    for (i = 0; i < h; i++) // height image
    {
        for (j = 0; j < w; j++) // width image
        {
            if(h_odata[i * w + j]!=MAX_BRIGHTNESS) {
                h_odata[i * w + j] = h_idata[i * w + j] / FADEDIV; // to obtain a faded background image
            }
        }
    }

    // free device memory
    hipFree(d_idata);
    hipFree(d_odata);
}

// print command line format
void usage(char *command) 
{
    printf("Usage: %s [-h] [-d device] [-i inputfile] [-o outputfile] [-r referenceFile] [-c th_count] [-t th_diff] [-m]\n", command);
}

// main
int main( int argc, char** argv) 
{

    // default command line options
    int deviceId = 0;
    char *fileIn        = (char *)"house.pgm",
         *fileOut       = (char *)"resultCuda.pgm",
         *referenceOut  = (char *)"referenceCuda.pgm";
    unsigned int th_count = 9, th_diff = 50, nonmaxflag = 0;

    // parse command line arguments
    int opt;
    while ((opt = getopt(argc, argv, "d:i:o:r:c:t:mh")) != -1)
    {
        switch(opt)
        {

            case 'd':
                if(sscanf(optarg,"%d",&deviceId)!=1)
                {
                    usage(argv[0]);
                    exit(1);
                }
                break;

            case 'i':
                if(strlen(optarg)==0)
                {
                    usage(argv[0]);
                    exit(1);
                }

                fileIn = strdup(optarg);
                break;
            case 'o':
                if(strlen(optarg)==0)
                {
                    usage(argv[0]);
                    exit(1);
                }
                fileOut = strdup(optarg);
                break;
            case 'r':
                if(strlen(optarg)==0)
                {
                    usage(argv[0]);
                    exit(1);
                }
                referenceOut = strdup(optarg);
                break;
            case 'c':
                if (strlen(optarg) == 0 || sscanf(optarg, "%d", &th_count) != 1)
                {
                    usage(argv[0]);
                    exit(1);
                }
                break;
            case 't':
                if (strlen(optarg) == 0 || sscanf(optarg, "%d", &th_diff) != 1)
                {
                    usage(argv[0]);
                    exit(1);
                }
                break;
            case 'm':
                nonmaxflag = 1;
                break;
            case 'h':
                usage(argv[0]);
                exit(0);
                break;

        }
    }

    // select cuda device
    checkCudaErrors( hipSetDevice( deviceId ) );
    
    // create events to measure host fast detector time and device fast detector time

    hipEvent_t startH, stopH, startD, stopD;
    checkCudaErrors( hipEventCreate(&startH) );
    checkCudaErrors( hipEventCreate(&stopH)  );
    checkCudaErrors( hipEventCreate(&startD) );
    checkCudaErrors( hipEventCreate(&stopD)  );



    // allocate host memory
    pixel_t * h_idata=NULL;
    unsigned int h,w;

    //load pgm
    if (sdkLoadPGM<pixel_t>(fileIn, &h_idata, &w, &h) != true) {
        printf("Failed to load image file: %s\n", fileIn);
        exit(1);
    }

    // allocate mem for the result on host side
    pixel_t * h_odata   = (pixel_t *) malloc( h*w*sizeof(pixel_t));
    pixel_t * reference = (pixel_t *) malloc( h*w*sizeof(pixel_t));
 
    // detect corners at host

    checkCudaErrors( hipEventRecord( startH, 0 ) );
    fastDetectorHost(h_idata, w, h, th_count, th_diff, nonmaxflag, reference);
    checkCudaErrors( hipEventRecord( stopH, 0 ) ); 
    checkCudaErrors( hipEventSynchronize( stopH ) );

    // detect corners at GPU
    checkCudaErrors( hipEventRecord( startD, 0 ) );
    fastDetectorDevice(h_idata, w, h, th_count, th_diff, nonmaxflag, h_odata);
    checkCudaErrors( hipEventRecord( stopD, 0 ) ); 
    checkCudaErrors( hipEventSynchronize( stopD ) );
    
    // check if kernel execution generated and error
    getLastCudaError("Kernel execution failed");

    float timeH, timeD;
    checkCudaErrors( hipEventElapsedTime( &timeH, startH, stopH ) );
    printf( "Host processing time: %f (ms)\n", timeH);
    checkCudaErrors( hipEventElapsedTime( &timeD, startD, stopD ) );
    printf( "Device processing time: %f (ms)\n", timeD);

    // save output images
    if (sdkSavePGM<pixel_t>(referenceOut, reference, w, h) != true) {
        printf("Failed to save image file: %s\n", referenceOut);
        exit(1);
    }
    if (sdkSavePGM<pixel_t>(fileOut, h_odata, w, h) != true) {
        printf("Failed to save image file: %s\n", fileOut);
        exit(1);
    }

    // cleanup memory
    free( h_idata);
    free( h_odata);
    free( reference);

    checkCudaErrors( hipDeviceReset() );
}
